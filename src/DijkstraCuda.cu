#include "hip/hip_runtime.h"
#include "DijkstraCuda.h"
#include <climits>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>

__global__ static void relaxNeighbors(int E, int u,
                                      const int* src,
                                      const int* dst,
                                      const int* w,
                                      int* dist) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= E) return;
  if (src[idx] == u) {
    int du = dist[u];
    if (du < INT_MAX) {
      int v  = dst[idx];
      int nd = du + w[idx];
      atomicMin(&dist[v], nd);
    }
  }
}

void DijkstraCuda::run(int source, std::vector<int>& h_dist) {

  int* d_dist = nullptr;
  CUDA_CHECK(hipMalloc(&d_dist, V * sizeof(int)));
  std::vector<int> hostDist(V, INT_MAX);
  hostDist[source] = 0;
  CUDA_CHECK(hipMemcpy(d_dist, hostDist.data(),
                        V * sizeof(int), hipMemcpyHostToDevice));

  std::vector<char> visited(V, 0);

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  const int threads = 65536;
  const int blocks  = (E + threads - 1) / threads;

  for (int iter = 0; iter < V; ++iter) {
    CUDA_CHECK(hipMemcpy(hostDist.data(), d_dist,
                          V * sizeof(int), hipMemcpyDeviceToHost));

    int u = -1, best = INT_MAX;
    for (int v = 0; v < V; ++v) {
      if (!visited[v] && hostDist[v] < best) {
        best = hostDist[v];
        u = v;
      }
    }
    if (u < 0 || best == INT_MAX) break;
    visited[u] = 1;

    relaxNeighbors<<<blocks, threads>>>(E, u, d_src, d_dst, d_w, d_dist);
    CUDA_CHECK(hipDeviceSynchronize());
  }

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  CUDA_CHECK(hipEventElapsedTime(&lastTimeMs, start, stop));

  CUDA_CHECK(hipMemcpy(h_dist.data(), d_dist,
                        V * sizeof(int), hipMemcpyDeviceToHost));

  hipFree(d_dist);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}
