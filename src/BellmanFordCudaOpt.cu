#include "hip/hip_runtime.h"
#include "BellmanFordCudaOpt.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <limits>
#include <algorithm>

#define INF 1000000000

// -----------------------------------------------------------------------------
// Kernel: one warp per vertex in the **current frontier**
// -----------------------------------------------------------------------------
__global__ void bf_frontier_warp(
    int                V,
    const int         *rowPtrIn,
    const int         *colIn,
    const int         *wIn,
    const int         *dist_cur,
          int         *dist_next,
    const int         *frontier_curr,
    int                frontier_size,
          int         *frontier_next,
          int         *frontier_next_size
) {
    int globalLane = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId     = globalLane >> 5;      // which warp
    int lane       = globalLane &  31;     // which lane in that warp
    if (warpId >= frontier_size) return;

    int v = frontier_curr[warpId];

    int start = rowPtrIn[v], end = rowPtrIn[v+1];
    int best = dist_cur[v];

    for (int e = start + lane; e < end; e += 32) {
        int u = colIn[e], w = wIn[e];
        int du = dist_cur[u];
        if (du != INF) {
            int cand = du + w;
            if (cand < best) best = cand;
        }
    }

    #pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        int other = __shfl_down_sync(0xFFFFFFFF, best, offset);
        best = min(best, other);
    }

    if (lane == 0) {
        int old = dist_cur[v];
        dist_next[v] = best;
        if (best < old) {
            int idx = atomicAdd(frontier_next_size, 1);
            frontier_next[idx] = v;
        }
    }
}

// -----------------------------------------------------------------------------
// Host: run() builds inbound‐CSR from the original arrays, then does
// the frontier‐driven warp kernel.
// -----------------------------------------------------------------------------
void BellmanFordCudaOpt::run(int source, std::vector<int>& dist) {
    if (V <= 0 || E <= 0) {
        std::cerr << "Error: Graph uninitialized!" << std::endl;
        return;
    }

    // --- 0) Copy original edge arrays back to host ---
    std::vector<int> h_src(E), h_dst(E), h_w(E);
    CUDA_CHECK(hipMemcpy(h_src.data(), d_src, E * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_dst.data(), d_dst, E * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_w.data(),   d_w,   E * sizeof(int), hipMemcpyDeviceToHost));

    // --- 1) Build inbound CSR on host ---
    std::vector<int> rowPtrIn(V+1, 0), colIn(E), wIn(E);

    // 1a) count incoming edges per vertex
    for (int i = 0; i < E; ++i) {
        ++rowPtrIn[ h_dst[i] + 1 ];
    }
    // 1b) prefix-sum
    for (int v = 1; v <= V; ++v) {
        rowPtrIn[v] += rowPtrIn[v-1];
    }
    // 1c) scatter into colIn/wIn
    std::vector<int> writePtr = rowPtrIn;
    for (int i = 0; i < E; ++i) {
        int v   = h_dst[i];
        int pos = writePtr[v]++;
        colIn[pos] = h_src[i];
        wIn[pos]   = h_w[i];
    }

    // --- 2) Upload CSR to the device ---
    int *d_rowPtrIn = nullptr, *d_colIn = nullptr, *d_wIn = nullptr;
    CUDA_CHECK(hipMalloc(&d_rowPtrIn, (V+1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_colIn,     E    * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_wIn,       E    * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_rowPtrIn, rowPtrIn.data(), (V+1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_colIn,    colIn.data(),    E    * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_wIn,      wIn.data(),      E    * sizeof(int), hipMemcpyHostToDevice));

    // --- 3) Allocate + init distance buffers (double‐buffer) ---
    int *d_dist_cur = nullptr, *d_dist_next = nullptr;
    CUDA_CHECK(hipMalloc(&d_dist_cur,  V * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_dist_next, V * sizeof(int)));
    std::vector<int> hostDist(V, INF);
    hostDist[source] = 0;
    CUDA_CHECK(hipMemcpy(d_dist_cur, hostDist.data(), V * sizeof(int), hipMemcpyHostToDevice));

    // --- 4) Allocate + init frontier buffers (start all active) ---
    std::vector<int> h_frontier(V);
    std::iota(h_frontier.begin(), h_frontier.end(), 0);
    int *d_frontier_curr = nullptr, *d_frontier_next = nullptr, *d_frontier_next_size = nullptr;
    CUDA_CHECK(hipMalloc(&d_frontier_curr,      V * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_frontier_next,      V * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_frontier_next_size,    sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_frontier_curr, h_frontier.data(), V * sizeof(int), hipMemcpyHostToDevice));
    int frontier_size = V;

    // --- 5) Main loop: warp-per-vertex over frontier, early exit ---
    hipEvent_t tic, toc;
    CUDA_CHECK(hipEventCreate(&tic));
    CUDA_CHECK(hipEventCreate(&toc));
    CUDA_CHECK(hipEventRecord(tic));

    const int WARPS_PER_BLOCK   = 4;
    const int THREADS_PER_BLOCK = WARPS_PER_BLOCK * 32;

    for (int iter = 0; iter < V - 1 && frontier_size > 0; ++iter) {
        // reset next-frontier counter
        CUDA_CHECK(hipMemset(d_frontier_next_size, 0, sizeof(int)));

        int blocks = (frontier_size + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
        bf_frontier_warp<<<blocks, THREADS_PER_BLOCK>>>(
            V,
            d_rowPtrIn, d_colIn, d_wIn,
            d_dist_cur, d_dist_next,
            d_frontier_curr, frontier_size,
            d_frontier_next, d_frontier_next_size
        );
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // fetch new frontier size
        CUDA_CHECK(hipMemcpy(&frontier_size, d_frontier_next_size, sizeof(int), hipMemcpyDeviceToHost));

        // swap buffers
        std::swap(d_dist_cur,       d_dist_next);
        std::swap(d_frontier_curr,  d_frontier_next);
    }

    CUDA_CHECK(hipEventRecord(toc));
    CUDA_CHECK(hipEventSynchronize(toc));
    CUDA_CHECK(hipEventElapsedTime(&lastTimeMs, tic, toc));

    // --- 6) Copy back final distances & clean up ---
    CUDA_CHECK(hipMemcpy(hostDist.data(), d_dist_cur, V * sizeof(int), hipMemcpyDeviceToHost));
    dist = hostDist;

    CUDA_CHECK(hipFree(d_rowPtrIn));
    CUDA_CHECK(hipFree(d_colIn));
    CUDA_CHECK(hipFree(d_wIn));
    CUDA_CHECK(hipFree(d_dist_cur));
    CUDA_CHECK(hipFree(d_dist_next));
    CUDA_CHECK(hipFree(d_frontier_curr));
    CUDA_CHECK(hipFree(d_frontier_next));
    CUDA_CHECK(hipFree(d_frontier_next_size));
    CUDA_CHECK(hipEventDestroy(tic));
    CUDA_CHECK(hipEventDestroy(toc));
}
