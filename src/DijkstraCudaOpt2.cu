#include "hip/hip_runtime.h"
#include "DijkstraCudaOpt2.h"
#include <climits>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <vector>
#include <queue>
#include <utility>
#include <functional>

#ifndef CUDA_CHECK
#define CUDA_CHECK(call)                                                     \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,    \
              hipGetErrorString(err));                                      \
      exit(EXIT_FAILURE);                                                     \
    }                                                                         \
  } while (0)
#endif

// Neighbor-relaxation kernel (same as original)
__global__ static void relaxNeighbors(int E, int u,
                                      const int* src,
                                      const int* dst,
                                      const int* w,
                                      int* dist) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= E) return;
    if (src[idx] == u) {
        int du = dist[u];
        if (du < INT_MAX) {
            int v  = dst[idx];
            int nd = du + w[idx];
            atomicMin(&dist[v], nd);
        }
    }
}

void DijkstraCudaOpt2::run(int source, std::vector<int>& h_dist) {
    // Allocate and initialize distance array on GPU
    int* d_dist = nullptr;
    CUDA_CHECK(hipMalloc(&d_dist, V * sizeof(int)));
    std::vector<int> hostDist(V, INT_MAX);
    hostDist[source] = 0;
    CUDA_CHECK(hipMemcpy(d_dist, hostDist.data(),
                          V * sizeof(int), hipMemcpyHostToDevice));

    // Allocate and initialize visited flags on host and device
    std::vector<char> visited(V, 0);
    unsigned char* d_visited = nullptr;
    CUDA_CHECK(hipMalloc(&d_visited, V * sizeof(unsigned char)));
    CUDA_CHECK(hipMemset(d_visited, 0, V * sizeof(unsigned char)));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    // Main Dijkstra loop using CPU priority queue for min selection
    const int TPB = 65536;
    int blocks = (E + TPB - 1) / TPB;

    for (int iter = 0; iter < V; ++iter) {
        // Copy updated distances back to host
        CUDA_CHECK(hipMemcpy(hostDist.data(), d_dist,
                              V * sizeof(int), hipMemcpyDeviceToHost));

        // Build a min-heap of (dist, vertex)
        using PII = std::pair<int,int>;
        std::priority_queue<PII, std::vector<PII>, std::greater<PII>> pq;
        for (int v = 0; v < V; ++v) {
            if (!visited[v]) pq.emplace(hostDist[v], v);
        }

        // Extract the smallest unvisited vertex
        int u = -1;
        int bestDist = INT_MAX;
        while (!pq.empty()) {
            auto [d, idx] = pq.top(); pq.pop();
            if (visited[idx]) continue;
            u = idx;
            bestDist = d;
            break;
        }
        if (u < 0 || bestDist == INT_MAX) break;

        // Mark visited on host and device
        visited[u] = 1;
        unsigned char one = 1;
        CUDA_CHECK(hipMemcpy(d_visited + u, &one,
                              sizeof(unsigned char),
                              hipMemcpyHostToDevice));

        // Relax neighbors of u on GPU
        relaxNeighbors<<<blocks, TPB>>>(E, u,
                                        d_src,
                                        d_dst,
                                        d_w,
                                        d_dist);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    // Record timing and clean up events
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&lastTimeMs, start, stop));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // Copy final distances back to host
    CUDA_CHECK(hipMemcpy(h_dist.data(), d_dist,
                          V * sizeof(int), hipMemcpyDeviceToHost));

    // Clean up GPU memory
    hipFree(d_dist);
    hipFree(d_visited);
}
